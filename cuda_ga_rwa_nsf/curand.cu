#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "util.cuh"

__device__ hiprandStateXORWOW_t state;
__device__ unsigned long long int BIGN = 2147483647;

__device__ double dRand(int max)
{
    //hiprandState state;
    //hiprandStateXORWOW_t state;
    int tId = (blockDim.x * blockIdx.x) + threadIdx.x;
    hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);

    return BIGN * hiprand_uniform(&state);
}

