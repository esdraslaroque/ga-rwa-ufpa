#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "util.cuh"

__device__ hiprandStateXORWOW_t state;

__device__ double dRand(int max)
{
    //hiprandState state;
    //hiprandStateXORWOW_t state;
    int tId = (blockDim.x * blockIdx.x) + threadIdx.x;
    hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);

    return -(max) * hiprand_uniform(&state);
}

