#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "util.cuh"

__device__ hiprandStateXORWOW_t state;

__device__ double dRand()
{
    unsigned long long int BIGNN = 2147483647;
    int tId = (blockDim.x * blockIdx.x) + threadIdx.x;
    hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);

    return (BIGNN+1) * hiprand_uniform(&state);
}

